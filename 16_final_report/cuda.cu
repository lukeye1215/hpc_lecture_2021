#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <chrono>
using namespace std;
//bash 01_run_cuda.sh
//N = 256, 80~ GFlops
//N = 512, 210~ GFlops
//N = 1024, 350~ GFlops

#define M 256 
//number of rows in the matrix 
// N >= M

__global__ void matmul(float *A, float *B, float *C, int N){
    int i = blockIdx.y ; 
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    float sum = 0;
    for (int k=0; k<N; k++){
        sum += A[N*i+k] * B[N*k+j]; 
    }
    C[N*i+j] = sum; 
}

int main(int argc, char **argv){
    int mpisize, mpirank;
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &mpisize);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpirank);
    
    //const int N = 512;
    const int N = 1024; 
    int matsize = N * N * sizeof(float);
    float *A, *B, *C;   
    hipMallocManaged(&A, matsize);
    hipMallocManaged(&B, matsize);
    hipMallocManaged(&C, matsize);

    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            A[N*i+j] = drand48();
            B[N*i+j] = drand48();
        }
    }
    
    double comp_time = 0, comm_time = 0;

    for(int irank=0; irank<mpisize; irank++) { // each rank 
        dim3 grid(N/M, N);
        auto tic = chrono::steady_clock::now();
        //offset = N/mpisize*((mpirank+irank) % mpisize);
        /*matrix multiplication*/
        /***
        for (int i=0; i<N/mpisize; i++)
            for (int j=0; j<N/mpisize; j++)
                for (int k=0; k<N; k++)
                    subC[N*i+j+offset] += subA[N*i+k] * subB[N/mpisize*k+j]; //subC[i, j] = subA[i, k] * subB[k, j]
        ***/  
        matmul<<<grid, M>>>(A, B, C, N); 
        // N/M >= 1 
        hipDeviceSynchronize();
        auto toc = chrono::steady_clock::now();
        comp_time += chrono::duration<double>(toc - tic).count();
        tic = chrono::steady_clock::now();
        comm_time += chrono::duration<double>(tic - toc).count();
  }

    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            for (int k=0; k<N; k++){
                C[N*i+j] -= A[N*i+k] * B[N*k+j];
            }
        }
    }
  
    double err = 0;
    
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            err += fabs(C[N*i+j]);
        }   
    }
    
    if(mpirank==0) {
        double time = comp_time+comm_time;
        printf("N    : %d\n",N);
        printf("comp : %lf s\n", comp_time);
        printf("comm : %lf s\n", comm_time);
        printf("total: %lf s (%lf GFlops)\n",time,2.*N*N*N/time/1e9);
        printf("error: %lf\n",err/N/N);
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);

    MPI_Finalize();
}
